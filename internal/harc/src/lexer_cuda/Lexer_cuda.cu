#include "hip/hip_runtime.h"
#include <harc/lexer_cuda/Lexer_cuda.hpp>
#include <harc/lexer/Tokenization.hpp>
#include <harc/lexer/Lexing_errors.hpp>
#include <harc/cuda_common/Codepoint_manipulation.cuh>

#include <cstdint>
#include <harc/Translation_unit_cuda.hpp>

namespace harc::cuda::tokenizer {

    const std::uint32_t no_codepoint = 0xffffffffu;
    const std::uint32_t invalid_codepoint = 0xfffffffeu;



    ///
    /// \param byte_ptr Pointer to 4 sequential bytes which may be part of a
    /// codepoint beginning at the specified address.
    /// \return Decoded Unicode codepoint. no_codepoint if byte_ptr points to a
    /// continuation byte. invalid_codepoint if encoding is invalid.
    __device__
    std::uint32_t decode_utf8_codepoint(const char* byte_ptr) {
        // Read all bytes which may potentially be part of a codepoint beginning
        // at byte_ptr
        std::uint32_t codepoint_bytes;
        memcpy(&codepoint_bytes, byte_ptr, 4);

        std::uint32_t first_byte = codepoint_bytes & 0xff;

        // Return byte if the leading bit isn't set
        if (first_byte < 128) {
            return first_byte;
        }

        std::uint32_t leading_ones_count = __clz(~first_byte);

        // Set ret to  -1 if the current byte is a continuation byte
        std::uint32_t codepoint = 0;
        if (leading_ones_count == 1) {
            codepoint = UINT32_MAX;
        }

        // Making for zeroing out any bytes which aren't occupied by the codepoint
        // beginning with this thread's bytes
        std::uint32_t occupied_bytes_mask = ((std::uint32_t(1) << (leading_ones_count * 8)) - 1) << 8;

        // Zero out any bytes which are occupied by other codepoints
        std::uint32_t codepoint_bits = codepoint_bytes & occupied_bytes_mask;

        std::uint32_t leading_byte_mask = (1 << (7 - leading_ones_count)) - 1;

        // TODO: Consider use of 8/16-bit dot product instructions. This could
        //  potentially be an alternative that shaves off a few instructions
        codepoint |= codepoint_bits & leading_byte_mask;
        codepoint |= (codepoint_bits >> 0x08) & 0x3f;
        codepoint |= (codepoint_bits >> 0x10) & 0x3f;
        codepoint |= (codepoint_bits >> 0x18) & 0x3f;

        if (codepoint > 0x10FFFF) {
            codepoint = invalid_codepoint;
        }

        // Check that shortest form is respected
        if (leading_ones_count == 2 && codepoint <= 0x80) {
            codepoint = invalid_codepoint;
        }

        if (leading_ones_count == 3 && codepoint <= 0x800) {
            codepoint = invalid_codepoint;
        }

        if (leading_ones_count == 4 && codepoint <= 0x10000) {
            codepoint = invalid_codepoint;
        }

        bool is_surrogate = 0xD800 <= codepoint && codepoint <= 0xDBFF;
        if (is_surrogate) {
            codepoint = invalid_codepoint;
        }

        //Mask containing the bits that would come from
        const std::uint32_t continuation_byte_indicators = 0xC0C0C0C0u;

        //Check if there are four consecutive continuation bytes, which would be invalid
        if ((codepoint_bytes & continuation_byte_indicators) == continuation_byte_indicators) {
            codepoint = invalid_codepoint;
        }

        return codepoint;
    }

    ///
    /// Decodes a buffer of utf-8 encoded text into Unicode codepoints
    ///
    /// \param source Pointer to buffer containing raw utf-8 encoded source code.
    /// \param codepoint_buffer Buffer into which codepoints will be placed.
    __global__
    void decode_codepoints(const char* source, std::uint32_t* codepoint_buffer) {
        std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;

        codepoint_buffer[index] = decode_utf8_codepoint(source + index);
    }

    ///
    ///
    /// \param x First codepoint
    /// \param y Second codepoint
    /// \param z Third codepoint
    /// \return Type of token
    __device__
    harc::Token_type identify_token(std::uint32_t x, std::uint32_t y, std::uint32_t z) {
        harc::Token_type ret = Token_type::NULL_TOKEN;
        if (x < 128) {

        } else {

        }

        return harc::Token_type::NULL_TOKEN;
    }

    __global__
    void tokenization_kernel_ascii(
        char* source_bytes,
        lex::Error* error_buffer,
        std::size_t error_buffer_length
    ) {
        //
    }

    __global__
    void tokenization_kernel_utf8(
        char* source_bytes,
        std::uint32_t* codepoint_buffer,
        lex::Error* error_buffer,
        std::size_t error_buffer_length
    ) {

    }

    ///
    /// \param code_units. Pointer to buffer containing utf8 code units
    /// \param requires_utf8 Output variable. 1 if the text uses codepoints greater than 127. 0 Otherwise
    __global__
    void requires_utf8_kernel(char* code_units, int* requires_utf8) {
        std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;

        std::int8_t byte = code_units[index];

        // Check if leading bit is set
        if (byte < 0) {
            *requires_utf8 = 1;
        }
    }

    void lex_ascii(
        dim3 grid,
        dim3 block,
        hipStream_t stream,
        Translation_unit& unit,
        std::uint32_t warp_size
    ) {
        hipEvent_t kernel_start;
        hipEvent_t kernel_stop;

        hipEventCreate(&kernel_start);
        hipEventCreate(&kernel_stop);

        // Allocate error buffer
        hipMallocAsync(&unit.device_error_buffer, unit.error_buffer_size, stream);


    }

    void lex_utf8(
        dim3 grid,
        dim3 block,
        hipStream_t stream,
        Translation_unit& unit,
        std::uint32_t warp_size
    ) {

        hipEvent_t kernel_start;
        hipEvent_t kernel_stop;

        hipEventCreate(&kernel_start);
        hipEventCreate(&kernel_stop);

        // Allocate codepoint buffer
        hipMallocAsync(
            &unit.device_codepoint_buffer,
            unit.codepoint_buffer_size,
            stream
        );

        hipMemsetAsync(
            unit.device_codepoint_buffer,
            0,
            unit.codepoint_buffer_size,
            stream
        );

        hipStreamSynchronize(stream);

        hipError_t err = hipError_t::hipSuccess;

        // Dispatch utf8 tokenization kernel
        err = hipEventRecord(kernel_start, stream);
        decode_codepoints<<<grid, block, 0, stream>>>(
            unit.device_source_buffer,
            unit.device_codepoint_buffer
        );
        err = hipEventRecord(kernel_stop, stream);

        hipStreamSynchronize(stream);

        err = hipEventElapsedTime(
            &unit.timings.tokenization_latency,
            kernel_start,
            kernel_stop
        );

        hipEventDestroy(kernel_start);
        hipEventDestroy(kernel_stop);

        hipStreamSynchronize(stream);
    }

    void lex(
        dim3 grid,
        dim3 block,
        hipStream_t stream,
        Translation_unit& unit,
        std::uint32_t warp_size
    ) {
        dim3 grid_size{};
        dim3 block_size{};

        if (unit.is_pure_ascii) {
            lex_ascii(grid, block, stream, unit, warp_size);
        } else {
            lex_utf8(grid, block, stream, unit, warp_size);
        }
    }

}
