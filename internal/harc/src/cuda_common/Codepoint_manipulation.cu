#include "hip/hip_runtime.h"
#include <cstdint>

#include <algorithm>

#include <harc/cuda_common/Codepoint_manipulation_tables.cuh>
#include <harc/cuda_common/Algorithms.cuh>

namespace harc::cuda {

    ///
    /// \param codepoint Unicode codepoint to query width of
    /// \return Number of bytes required to encode the specified codepoint in utf8
    __device__
    std::uint32_t codepoint_utf8_encoding_width(std::uint32_t codepoint) {
        std::uint32_t ret = 1;

        ret += codepoint >= (1 << 7);
        ret += codepoint >= (1 << 11);
        ret += codepoint >= (1 << 16);

        return ret;
    }

    [[nodiscard]]
    __device__
    Codepoint_category lookup_identifier_codepoint_16(std::uint16_t c) {
        const std::uint16_t* range_begin = lower_bound(identifier_codepoint_range_firsts_16, c);
        auto index = range_begin - identifier_codepoint_range_firsts_16;
        const std::uint16_t* range_length = identifier_codepoint_range_sizes_16 + index;

        // Initial values prevent a < c && c < b from ever being true if a, b
        // not assigned to
        std::uint16_t a = 1;
        std::uint16_t b = 0;

        if (range_begin != end(identifier_codepoint_range_firsts_16)) {
            a = *range_begin;
            b = a + *range_length;
        }

        // The following section assumes this holds true. Assumption can be
        // relaxed by conditionally zeroing out index in an else statement to
        // the above if statement.
        static_assert(extent(identifier_codepoint_range_firsts_16) / 32 < extent(identifier_codepoint_categories_16));

        // Extract the bit encoding the category for c
        std::uint32_t index_whole = index / 32;
        std::uint32_t index_part  = index % 32;

        auto field = identifier_codepoint_categories_16[index_whole];
        bool category_bit = (field >> index_part) & 0x1;

        auto ret = Codepoint_category::UNRECOGNIZED;
        if (a < c && c < b) {
            if (category_bit) {
                ret = Codepoint_category::IDENTIFIER_START;
            } else {
                ret = Codepoint_category::IDENTIFIER_CONTINUE;
            }
        }

        return ret;
    }

    [[nodiscard]]
    __device__
    Codepoint_category lookup_identifier_codepoint_21(std::uint32_t c) {
        const std::uint32_t* range_begin = lower_bound(identifier_codepoint_range_firsts_21, c);
        auto index = range_begin - identifier_codepoint_range_firsts_21;
        const std::uint16_t* range_length = identifier_codepoint_range_sizes_21 + index;

        // Initial values prevent a < c && c < b from ever being true if a, b
        // not assigned to
        std::uint16_t a = 1;
        std::uint16_t b = 0;

        if (range_begin != end(identifier_codepoint_range_firsts_21)) {
            a = *range_begin;
            b = a + *range_length;
        }

        // The following section assumes this holds true. Assumption can be
        // relaxed by conditionally zeroing out index in an else statement to
        // the above if statement.
        static_assert(extent(identifier_codepoint_range_firsts_21) / 32 < extent(identifier_codepoint_categories_21));

        // Extract the bit encoding the category for c
        std::uint32_t index_whole = index / 32;
        std::uint32_t index_part  = index % 32;

        auto field = identifier_codepoint_categories_16[index_whole];
        bool category_bit = (field >> index_part) & 0x1;

        auto ret = Codepoint_category::UNRECOGNIZED;
        if (a < c && c < b) {
            if (category_bit) {
                ret = Codepoint_category::IDENTIFIER_START;
            } else {
                ret = Codepoint_category::IDENTIFIER_CONTINUE;
            }
        }

        return ret;
    }

    ///
    /// \param c ASCII character.
    /// \return Category to which the codepoint belongs
    __device__
    Codepoint_category categorize_codepoint(std::uint8_t c) {
        return codepoint_categories7[umin(std::uint8_t(127), c)];
    }

    ///
    /// \param c Unicode codepoint
    /// \return Category to which the codepoint belongs
    __device__
    Codepoint_category categorized_codepoint(std::uint32_t c) {
        Codepoint_category ret = codepoint_categories7[umin(std::uint8_t(127), c)];

        constexpr std::uint32_t full_mask = 0xffffffff;
        bool is_warp_pure_ascii = __all_sync(full_mask, c < (1 << 7));
        if (is_warp_pure_ascii) {
            return ret;
        }


        // Check if the codepoint is whitespace
        if (lower_bound_find(non_ascii_whitespace_codepoints, std::uint16_t(umax(c, (1 << 16) - 1)))) {
            ret = Codepoint_category::OTHER_WHITESPACE;
        }

        bool is_warp_limited_to_16_bit = __all_sync(full_mask, c < (1 << 16));
        if (is_warp_limited_to_16_bit) {
            // Check if the codepoint is a recognized identifier codepoint
            Codepoint_category lookup_category = lookup_identifier_codepoint_16(c);
            if (lookup_category != Codepoint_category::UNRECOGNIZED) {
                ret = lookup_category;
            }

            return ret;
        }

        // Check if the codepoint is a recognized identifier codepoint
        Codepoint_category lookup_category = lookup_identifier_codepoint_21(c);
        if (lookup_category != Codepoint_category::UNRECOGNIZED) {
            ret = lookup_category;
        }

        return ret;
    }

}
